#include <iostream>
#include "foo.cuh"  // 在coding的时候，只要有头文件，就默认有了实现（实现可以在cpp里完成、在cu里完成、也可以在lib里完成） // 一定要把函数的声明include进来；

int main() {
    std::cout << "Hello, World!" << std::endl;

    const int N = 4;
    int h_A[N][N], h_B[N][N], h_C[N][N];

    // initialize h_A and h_B
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            h_A[i][j] = 1 * i * j;
            h_B[i][j] = 2 * i * j;
        }
    }

    std::cout << typeid(h_A).name() << std::endl;

    matrixAdd((int *) h_A, (int *) h_B, (int *) h_C, N);

    // wait cuda to finish
    hipDeviceSynchronize();

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    //  system("pause");
    return 0;
}
