
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void thread_idx_1()
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int id = blockDim.x * bid + tid;
    printf("blockId : %d, threadId : %d, id : %d\n", bid, tid, id);
}

int main(void)
{
    thread_idx_1<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}
