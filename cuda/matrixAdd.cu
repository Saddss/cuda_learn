
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_add(float* p1, float* p2, float* p3, int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int id = tid + bid * blockDim.x;
    if (id < N) {
        p3[id] = p1[id] + p2[id];
    } else {
        printf("Thread %d is out of bounds!\n", id);  // 这可能会帮助你看到超出范围的线程
    }
}

void initData(float* p, int elementNum)
{
    for (int i = 0; i < elementNum; i++)
    {
        p[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

int main()
{
    // 检测计算机GPU数量
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount);

    if (error != hipSuccess || iDeviceCount == 0){
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    else{
        printf("The count of GPUs is %d.\n", iDeviceCount);
    }

    // 设置执行
    int iDev = 0;
    error = hipSetDevice(iDev);
    if (error != hipSuccess){
        printf("fail to set GPU 0 for computing.\n");
        exit(-1);
    }
    else{
        printf("set GPU 0 for computing.\n");
    }

    int iElemCount = 513;
    int byteCount = iElemCount * sizeof(float);
    float* p1 = (float*)malloc(byteCount);
    float* p2 = (float*)malloc(byteCount);
    float* p3 = (float*)malloc(byteCount);

    memset(p1, 0, byteCount);
    memset(p2, 0, byteCount);
    memset(p3, 0, byteCount);

    initData(p1, iElemCount);
    initData(p2, iElemCount);

    float* dp1, *dp2, *dp3;
    hipMalloc((float**)&dp1, byteCount);
    hipMalloc((float**)&dp2, byteCount);
    hipMalloc((float**)&dp3, byteCount);

    hipMemcpy(dp1, p1, byteCount, hipMemcpyHostToDevice);
    hipMemcpy(dp2, p2, byteCount, hipMemcpyHostToDevice);
    hipMemcpy(dp3, p3, byteCount, hipMemcpyHostToDevice);
    
    dim3 block(32);
    dim3 grid((iElemCount + block.x - 1) / 32);
    kernel_add<<<grid, block>>>(dp1, dp2, dp3, iElemCount);
    hipDeviceSynchronize();

    hipMemcpy(p1, dp1, byteCount, hipMemcpyDeviceToHost);
    hipMemcpy(p2, dp2, byteCount, hipMemcpyDeviceToHost);
    hipMemcpy(p3, dp3, byteCount, hipMemcpyDeviceToHost);

    for (int i = 0; i < iElemCount; i++)
    {
        printf("%.2f + %.2f = %.2f\n", p1[i], p2[i], p3[i]);
    }

    free(p1);
    free(p2);
    free(p3);
    hipFree(dp1);
    hipFree(dp2);
    hipFree(dp3);
    return 0;
}