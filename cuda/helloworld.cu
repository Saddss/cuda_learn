
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void helloWorld()
{
    printf("Hello World from gpu!\n");
}

int main()
{
    helloWorld<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}