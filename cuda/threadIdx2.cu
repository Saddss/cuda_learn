
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void thread_idx_2()
{
    const int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    const int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    const int id = blockId * (blockDim.x * blockDim.y) + threadId;
    printf("blockId: %d, threadId: %d, id: %d\n", blockId, threadId, id);
}

int main()
{
    dim3 gridDim(2, 4);
    dim3 blockDim(5, 2);
    thread_idx_2<<<gridDim, blockDim>>>();
    return 0;
}