#include "hip/hip_runtime.h"
#include "foo.cuh"
#include "hip/hip_runtime.h"
#include "stdio.h"  //函数在实现的时候，要用到很多依赖，在这里include进来！

__global__ void addKernel(int *A, int *B, int *C, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    printf("i is %d, j is %d\n", i, j);

    if (i < N && j < N) {
        C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

void matrixAdd(int *A, int *B, int *C, int N) {
    const int blockSize = 4;
    int TILE_WIDTH = 2; //
    dim3 dimGrid(N / TILE_WIDTH, N / TILE_WIDTH);  //定义一个Grid有多少个Block；

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);  //定义一个Blcok有多少个线程；

    int *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, N * N * sizeof(int));
    hipMalloc((void **) &d_B, N * N * sizeof(int));
    hipMalloc((void **) &d_C, N * N * sizeof(int));

    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);
    addKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}